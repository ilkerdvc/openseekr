#include "hip/hip_runtime.h"
/*
   Copyright 2019 by Andy Stokely
	All rights reserved
   Planar Milestone CUDA Code for the OpenMM Plugin SEEKR
*/

//path = /plugin/platforms/cuda/src/kernels

#define ELEM_SWAP(a,b) { int t=(a);(a)=(b);(b)=t; }

// TODO: these arguments might need to be arranged with arrays first, scalars last...

extern "C" __global__ void monitorPlanarMilestones(
                            const real4* __restrict__ posq,             // positions and charges
                            const mixed4* __restrict__ velm,             // velocities and masses
                            //const float* __restrict__ masses,           // the masses of all of the atoms
                            const int* __restrict__ numIndices1,         // number of atoms in receptor
                            const int* __restrict__ numIndices2,         // number of atoms in ligand
                            const float* __restrict__ length1,           // length of inner planar milestone
                            const float* __restrict__ length2,           // length of middle planar milestone ANDY
                            const float* __restrict__ length3,           // length of outer planar milestone
                            const int* __restrict__ atomIndices1,       // atom indices of receptor
                            const int* __restrict__ atomIndices2,       // atom indices of ligand
                            const int2* __restrict__ atomBounds1,
                            const int2* __restrict__ atomBounds2,
                            float* __restrict__ returncode,              // whether the milestone was crossed: 0 = uncrossed, 1 = crossed inner, 2 = crossed outer
                            float4* __restrict__ old_com1,              // Keeps track of the previous timestep's receptor COM to determine if the middle milestone was crossed
                            float4* __restrict__ old_com2,              // old ligand COM
                            const int numPlanarMilestones) {           // length of outer planar milestone

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
   //													DIDN'T CHANGE ANY OF THIS CODE
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    for (int index=blockIdx.x*blockDim.x+threadIdx.x; index<numPlanarMilestones; index+=blockDim.x*gridDim.x) { // Dedicate a given GPU to the milestone object
        //int atomid;
        returncode[index] = 0; // Initialize a nominal message to return to the CPU regarding milestone crossing
        
        real4 com1; // center of mass of the 'receptor'
        real4 com2; // center of mass of the 'ligand'
        float totalmass1 = 0.0; // total mass of the 'receptor' selection
        float totalmass2 = 0.0; // total mass of the 'ligand' selection
        float mass = 0.0; // temporary placeholder for masses of particles
        int i;
        int atomIndex; // temporary placeholder for atom index
        com1.x = com1.y = com1.z = 0.0;
        com2.x = com2.y = com2.z = 0.0;  // initialize the center of mass positions to 0,0,0
        
        for (i=atomBounds1[index].x; i<atomBounds1[index].y; i++) { // loop through this milestone set's atom indices
            atomIndex = atomIndices1[i]; // obtain this atom's index
            mass = 1.0 / velm[atomIndex].w; // extract the mass from the velm array
            com1 += posq[atomIndex]*mass; // add this atom's contribution to the center of mass
            totalmass1 += mass; // increment the selection's total mass by the particle's mass
        }
        com1 = com1 / totalmass1; // divide by total mass of all particles to get the normalized center of mass
        
        for (i=atomBounds2[index].x; i<atomBounds2[index].y; i++) { // see above for loop: applies to the ligand
            atomIndex = atomIndices2[i];
            mass = 1.0 / velm[atomIndex].w;
            com2 += posq[atomIndex]*mass;
            totalmass2 += mass;
        }
        com2 = com2 / totalmass2; 
        //old_com2 = old_com2 / totalmass2;
        
        // TODO: find a less hacky way to deal with this
        if (old_com1[index].x == -9.0e5) { // then this is the first step, so initialize old_posq to equal posq 
          returncode[index] = 4; //TODO **NO CHANGE***
          old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z; 
          old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z; 
        }

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


                
        real delta = com2.z - com1.z; // ANDY: com2.z - com1.z TODO **DONE** 
        float old_delta = old_com2[index].z - old_com1[index].z; // ANDY: use .z here too TODO **DONE** 
       
        old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z; // save current COM to be the next timestep's old_COM TODO **NO CHANGE**
        old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z; // TODO **NO CHANGE***
        
        if (delta < length1) { // crossed inner milestone ANDY: change length1 to z1 TODO **DONE**
            returncode[index] = 1;			
        } 
        else if ((delta - length2[index]*(old_delta - length2[index]) < 0) { // This will return true if the particle has crossed the middle milestone since the last timestep ANDY TODO **DONE**
            returncode[index] = 2;
        }
        else if (DELTA > length3[index]) { // crossed outer milestone ANDY TODO **DONE**
            returncode[index] = 3;
        
        }
        
        //returncode[index] = length2[index]*length2[index];
    }
}
0


























