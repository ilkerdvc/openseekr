#include "hip/hip_runtime.h"
/*
   Copyright 2018 by Lane Votapka
   All rights reserved
*/

#define ELEM_SWAP(a,b) { int t=(a);(a)=(b);(b)=t; }

// TODO: these arguments might need to be arranged with arrays first, scalars last...

extern "C" __global__ void monitorSphericalMilestones(
                            const real4* __restrict__ posq,             // positions and charges
                            const mixed4* __restrict__ velm,             // velocities and masses
                            //const float* __restrict__ masses,           // the masses of all of the atoms
                            const int* __restrict__ numIndices1,         // number of atoms in receptor
                            const int* __restrict__ numIndices2,         // number of atoms in ligand
                            const float* __restrict__ radius1,           // radius of inner spherical milestone
                            const float* __restrict__ radius2,           // radius of middle spherical milestone
                            const float* __restrict__ radius3,           // radius of outer spherical milestone
                            const int* __restrict__ atomIndices1,       // atom indices of receptor
                            const int* __restrict__ atomIndices2,       // atom indices of ligand
                            const int2* __restrict__ atomBounds1,
                            const int2* __restrict__ atomBounds2,
                            float* __restrict__ returncode,              // whether the milestone was crossed: 0 = uncrossed, 1 = crossed inner, 2 = crossed outer
                            float4* __restrict__ old_com1,
                            float4* __restrict__ old_com2,
                            const int numSphericalMilestones) {           // radius of outer spherical milestone
    
    for (int index=blockIdx.x*blockDim.x+threadIdx.x; index<numSphericalMilestones; index+=blockDim.x*gridDim.x) {
        //int atomid;
        returncode[index] = 0;
        
        real4 com1;
        real4 com2;
        float totalmass1 = 0.0;
        float totalmass2 = 0.0;
        float mass = 0.0;
        int i;
        int atomIndex;
        com1.x = com1.y = com1.z = 0.0;
        com2.x = com2.y = com2.z = 0.0;  // initialize the center of mass positions to 0,0,0
        
        for (i=atomBounds1[index].x; i<atomBounds1[index].y; i++) {
            atomIndex = atomIndices1[i];
            mass = 1.0 / velm[atomIndex].w;
            com1 += posq[atomIndex]*mass;
            totalmass1 += mass;
        }
        com1 = com1 / totalmass1;
        
        for (i=atomBounds2[index].x; i<atomBounds2[index].y; i++) {
            atomIndex = atomIndices2[i];
            mass = 1.0 / velm[atomIndex].w;
            com2 += posq[atomIndex]*mass;
            totalmass2 += mass;
        }
        com2 = com2 / totalmass2;
        //old_com2 = old_com2 / totalmass2;
        
        if (old_com1[index].x == -9.0e5) { // then this is the first step, so initialize old_posq to equal posq
          returncode[index] = 4;
          old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z;
          old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z;
        }
                
        real4 delta = com2 - com1;
        float4 old_delta = old_com2[index] - old_com1[index];
        real distSquared = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
        real old_distSquared = old_delta.x*old_delta.x + old_delta.y*old_delta.y + old_delta.z*old_delta.z;
        //real r = SQRT(distSquared);
        
        old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z;
        old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z;
        
        if (distSquared < radius1[index]*radius1[index]) { // crossed inner milestone
            returncode[index] = 1;
        } 
        else if ((distSquared - radius2[index]*radius2[index])*(old_distSquared - radius2[index]*radius2[index]) < 0) { // This will return true if the particle has crossed the middle milestone since the last timestep
            returncode[index] = 2;
        }
        else if (distSquared > radius3[index]*radius3[index]) { // crossed outer milestone
            returncode[index] = 3;
        
        }
        
        if (isnan(distSquared) == true) {
            returncode[index] = 5;
        }
        
        //returncode[index] = distSquared;
    }
}

/*
extern "C" __global__ void computeDistRest(
                            const real4* __restrict__ posq,             // positions and charges
                            const int2* __restrict__ atomIndices,       // pair of atom indices
                            const float4* __restrict__ distanceBounds,  // r1, r2, r3, r4
                            const float* __restrict__ forceConstants,   // k
                            int* __restrict__ indexToGlobal,            // array of indices into global arrays
                            float* __restrict__ energies,               // global array of restraint energies
                            float3* __restrict__ forceBuffer,           // temporary buffer to hold the force
                            const int numRestraints) {
    for (int index=blockIdx.x*blockDim.x+threadIdx.x; index<numRestraints; index+=blockDim.x*gridDim.x) {
        // get my global index
        const int globalIndex = indexToGlobal[index];

        // get the distances
        const float r1 = distanceBounds[index].x;
        const float r2 = distanceBounds[index].y;
        const float r3 = distanceBounds[index].z;
        const float r4 = distanceBounds[index].w;

        // get the force constant
        const float k = forceConstants[index];

        // get atom indices and compute distance
        int atomIndexA = atomIndices[index].x;
        int atomIndexB = atomIndices[index].y;
        real4 delta = posq[atomIndexA] - posq[atomIndexB];
        real distSquared = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
        real r = SQRT(distSquared);

        // compute force and energy
        float energy = 0.0;
        float dEdR = 0.0;
        float diff = 0.0;
        float diff2 = 0.0;
        float3 f;

        if(r < r1) {
            energy = k * (r - r1) * (r1 - r2) + 0.5 * k * (r1 - r2) * (r1 - r2);
            dEdR = k * (r1 - r2);
        }
        else if(r < r2) {
            diff = r - r2;
            diff2 = diff * diff;
            energy = 0.5 * k * diff2;
            dEdR = k * diff;
        }
        else if(r < r3) {
            dEdR = 0.0;
            energy = 0.0;
        }
        else if(r < r4) {
            diff = r - r3;
            diff2 = diff * diff;
            energy = 0.5 * k * diff2;
            dEdR = k * diff;
        }
        else {
            energy = k * (r - r4) * (r4 - r3) + 0.5 * k * (r4 - r3) * (r4 - r3);
            dEdR = k * (r4 - r3);
        }

        // store force into local buffer
        if (r > 0) {
            f.x = delta.x * dEdR / r;
            f.y = delta.y * dEdR / r;
            f.z = delta.z * dEdR / r;
        } else {
            f.x = 0.0;
            f.y = 0.0;
            f.z = 0.0;
        }
        forceBuffer[index] = f;

        // store energy into global buffer
        energies[globalIndex] = energy;
    }
}
*/
