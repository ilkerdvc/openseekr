#include "hip/hip_runtime.h"
/*
   Copyright 2019 by Andy Stokely and Lane Votapka
	All rights reserved
   PlanarZ Milestone CUDA Code for the OpenMM Plugin SEEKR
*/

//path = /plugin/platforms/cuda/src/kernels

#define ELEM_SWAP(a,b) { int t=(a);(a)=(b);(b)=t; }

// TODO: these arguments might need to be arranged with arrays first, scalars last...

extern "C" __global__ void monitorPlanarZMilestones(
                            const real4* __restrict__ posq,             // positions and charges
                            const mixed4* __restrict__ velm,             // velocities and masses
                            //const float* __restrict__ masses,           // the masses of all of the atoms
                            const int* __restrict__ numIndices1,         // number of atoms in receptor
                            const int* __restrict__ numIndices2,         // number of atoms in ligand
                            const float* __restrict__ offset1,           // offset of inner planarZ milestone
                            const float* __restrict__ offset2,           // offset of middle planarZ milestone ANDY
                            const float* __restrict__ offset3,           // offset of outer planarZ milestone
                            const int* __restrict__ atomIndices1,       // atom indices of receptor
                            const int* __restrict__ atomIndices2,       // atom indices of ligand
                            const int2* __restrict__ atomBounds1,
                            const int2* __restrict__ atomBounds2,
                            float* __restrict__ returncode,              // whether the milestone was crossed: 0 = uncrossed, 1 = crossed inner, 2 = crossed outer
                            float4* __restrict__ old_com1,              // Keeps track of the previous timestep's receptor COM to determine if the middle milestone was crossed
                            float4* __restrict__ old_com2,              // old ligand COM
                            const int numPlanarZMilestones) {           // offset of outer planarZ milestone

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
   //													DIDN'T CHANGE ANY OF THIS CODE
  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    for (int index=blockIdx.x*blockDim.x+threadIdx.x; index<numPlanarZMilestones; index+=blockDim.x*gridDim.x) { // Dedicate a given GPU to the milestone object
        //int atomid;
        returncode[index] = 0; // Initialize a nominal message to return to the CPU regarding milestone crossing
        
        real4 com1; // center of mass of the 'receptor'
        real4 com2; // center of mass of the 'ligand'
        float totalmass1 = 0.0; // total mass of the 'receptor' selection
        float totalmass2 = 0.0; // total mass of the 'ligand' selection
        float mass = 0.0; // temporary placeholder for masses of particles
        int i;
        int atomIndex; // temporary placeholder for atom index
        com1.x = com1.y = com1.z = 0.0;
        com2.x = com2.y = com2.z = 0.0;  // initialize the center of mass positions to 0,0,0
        
        for (i=atomBounds1[index].x; i<atomBounds1[index].y; i++) { // loop through this milestone set's atom indices
            atomIndex = atomIndices1[i]; // obtain this atom's index
            mass = 1.0 / velm[atomIndex].w; // extract the mass from the velm array
            com1 += posq[atomIndex]*mass; // add this atom's contribution to the center of mass
            totalmass1 += mass; // increment the selection's total mass by the particle's mass
        }
        com1 = com1 / totalmass1; // divide by total mass of all particles to get the normalized center of mass
        
        for (i=atomBounds2[index].x; i<atomBounds2[index].y; i++) { // see above for loop: applies to the ligand
            atomIndex = atomIndices2[i];
            mass = 1.0 / velm[atomIndex].w;
            com2 += posq[atomIndex]*mass;
            totalmass2 += mass;
        }
        com2 = com2 / totalmass2; 
        //old_com2 = old_com2 / totalmass2;
        
        // TODO: find a less hacky way to deal with this
        if (old_com1[index].x == -9.0e5) { // then this is the first step, so initialize old_posq to equal posq 
          returncode[index] = 4; //TODO **NO CHANGE***
          old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z; 
          old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z; 
        }

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


                
        real delta = com2.z - com1.z; // ANDY: com2.z - com1.z TODO **DONE** 
        float old_delta = old_com2[index].z - old_com1[index].z; // ANDY: use .z here too TODO **DONE** 
       
        old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z; // save current COM to be the next timestep's old_COM TODO **NO CHANGE**
        old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z; // TODO **NO CHANGE***
        
        if (delta < offset1) { // crossed inner milestone ANDY: change offset1 to z1 TODO **DONE**
            returncode[index] = 1;			
        } 
        else if ((delta - offset2[index]*(old_delta - offset2[index]) < 0) { // This will return true if the particle has crossed the middle milestone since the last timestep ANDY TODO **DONE**
            returncode[index] = 2;
        }
        else if (DELTA > offset3[index]) { // crossed outer milestone ANDY TODO **DONE**
            returncode[index] = 3;
        
        }
        
        //returncode[index] = offset2[index]*offset2[index];
    }
}

extern "C" __global__ void monitorSphericalMilestones(
                            const real4* __restrict__ posq,             // positions and charges
                            const mixed4* __restrict__ velm,             // velocities and masses
                            //const float* __restrict__ masses,           // the masses of all of the atoms
                            const int* __restrict__ numIndices1,         // number of atoms in receptor
                            const int* __restrict__ numIndices2,         // number of atoms in ligand
                            const float* __restrict__ radius1,           // radius of inner spherical milestone
                            const float* __restrict__ radius2,           // radius of middle spherical milestone
                            const float* __restrict__ radius3,           // radius of outer spherical milestone
                            const int* __restrict__ atomIndices1,       // atom indices of receptor
                            const int* __restrict__ atomIndices2,       // atom indices of ligand
                            const int2* __restrict__ atomBounds1,
                            const int2* __restrict__ atomBounds2,
                            float* __restrict__ returncode,              // whether the milestone was crossed: 0 = uncrossed, 1 = crossed inner, 2 = crossed outer
                            float4* __restrict__ old_com1,
                            float4* __restrict__ old_com2,
                            const int numSphericalMilestones) {           // radius of outer spherical milestone
    
    for (int index=blockIdx.x*blockDim.x+threadIdx.x; index<numSphericalMilestones; index+=blockDim.x*gridDim.x) {
        //int atomid;
        returncode[index] = 0;
        
        real4 com1;
        real4 com2;
        float totalmass1 = 0.0;
        float totalmass2 = 0.0;
        float mass = 0.0;
        int i;
        int atomIndex;
        com1.x = com1.y = com1.z = 0.0;
        com2.x = com2.y = com2.z = 0.0;  // initialize the center of mass positions to 0,0,0
        
        for (i=atomBounds1[index].x; i<atomBounds1[index].y; i++) {
            atomIndex = atomIndices1[i];
            mass = 1.0 / velm[atomIndex].w;
            com1 += posq[atomIndex]*mass;
            totalmass1 += mass;
        }
        com1 = com1 / totalmass1;
        
        for (i=atomBounds2[index].x; i<atomBounds2[index].y; i++) {
            atomIndex = atomIndices2[i];
            mass = 1.0 / velm[atomIndex].w;
            com2 += posq[atomIndex]*mass;
            totalmass2 += mass;
        }
        com2 = com2 / totalmass2;
        //old_com2 = old_com2 / totalmass2;
        
        if (old_com1[index].x == -9.0e5) { // then this is the first step, so initialize old_posq to equal posq
          returncode[index] = 4;
          old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z;
          old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z;
        }
                
        real4 delta = com2 - com1;
        float4 old_delta = old_com2[index] - old_com1[index];
        real distSquared = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
        real old_distSquared = old_delta.x*old_delta.x + old_delta.y*old_delta.y + old_delta.z*old_delta.z;
        //real r = SQRT(distSquared);
        
        old_com1[index].x = com1.x; old_com1[index].y = com1.y; old_com1[index].z = com1.z;
        old_com2[index].x = com2.x; old_com2[index].y = com2.y; old_com2[index].z = com2.z;
        
        if (distSquared < radius1[index]*radius1[index]) { // crossed inner milestone
            returncode[index] = 1;
        } 
        else if ((distSquared - radius2[index]*radius2[index])*(old_distSquared - radius2[index]*radius2[index]) < 0) { // This will return true if the particle has crossed the middle milestone since the last timestep
            returncode[index] = 2;
        }
        else if (distSquared > radius3[index]*radius3[index]) { // crossed outer milestone
            returncode[index] = 3;
        
        }
        
        //returncode[index] = radius2[index]*radius2[index];
    }
}



























